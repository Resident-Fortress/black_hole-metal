#include "hip/hip_runtime.h"
#include "../include/black_hole_cuda.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>

// Device constants
__constant__ float d_SagA_rs = SAGA_RS;
__constant__ float d_D_LAMBDA = D_LAMBDA;
__constant__ float d_ESCAPE_R = ESCAPE_R;

// Device utility functions
__device__ inline float3 make_float3_from_glm(const glm::vec3& v) {
    return make_float3(v.x, v.y, v.z);
}

__device__ inline float length_f3(const float3& v) {
    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ inline float3 normalize_f3(const float3& v) {
    float len = length_f3(v);
    return make_float3(v.x / len, v.y / len, v.z / len);
}

// Enhanced blackbody spectrum calculation
__device__ float3 blackbodySpectrum(float temperature) {
    // Simplified blackbody radiation using Planck's law approximation
    float r = fmaxf(0.0f, fminf(1.0f, 1.0f - expf(-6000.0f / temperature)));
    float g = fmaxf(0.0f, fminf(1.0f, 1.0f - expf(-4000.0f / temperature)));
    float b = fmaxf(0.0f, fminf(1.0f, 1.0f - expf(-2000.0f / temperature)));
    return make_float3(r, g, b);
}

// Gravitational redshift calculation
__device__ float gravitationalRedshift(float r) {
    return sqrtf(fmaxf(0.1f, 1.0f - d_SagA_rs / r));
}

// Doppler shift simulation
__device__ float3 dopplerShift(const float3& color, float velocity) {
    float factor = 1.0f + velocity * 0.3f;
    return make_float3(color.x * factor, color.y * factor, color.z * factor);
}

// Initialize ray from camera parameters
__device__ CudaRay initRay(const float3& pos, const float3& dir) {
    CudaRay ray;
    ray.x = pos.x;
    ray.y = pos.y; 
    ray.z = pos.z;
    ray.r = length_f3(pos);
    ray.theta = acosf(pos.z / ray.r);
    ray.phi = atan2f(pos.y, pos.x);

    // Calculate derivatives
    ray.dr = sinf(ray.theta) * cosf(ray.phi) * dir.x + 
             sinf(ray.theta) * sinf(ray.phi) * dir.y + 
             cosf(ray.theta) * dir.z;
    ray.dtheta = (cosf(ray.theta) * cosf(ray.phi) * dir.x + 
                  cosf(ray.theta) * sinf(ray.phi) * dir.y - 
                  sinf(ray.theta) * dir.z) / ray.r;
    ray.dphi = (-sinf(ray.phi) * dir.x + cosf(ray.phi) * dir.y) / 
               (ray.r * sinf(ray.theta));

    // Calculate conserved quantities
    ray.L = ray.r * ray.r * sinf(ray.theta) * ray.dphi;
    float f = 1.0f - d_SagA_rs / ray.r;
    float dt_dL = sqrtf((ray.dr * ray.dr) / f + 
                        ray.r * ray.r * (ray.dtheta * ray.dtheta + 
                        sinf(ray.theta) * sinf(ray.theta) * ray.dphi * ray.dphi));
    ray.E = f * dt_dL;

    return ray;
}

// Geodesic equation right-hand side
__device__ void geodesicRHS(const CudaRay& ray, float3& d1, float3& d2) {
    float r = ray.r;
    float theta = ray.theta;
    float dr = ray.dr;
    float dtheta = ray.dtheta;
    float dphi = ray.dphi;
    float f = 1.0f - d_SagA_rs / r;
    float dt_dL = ray.E / f;

    d1 = make_float3(dr, dtheta, dphi);
    
    d2.x = -(d_SagA_rs / (2.0f * r * r)) * f * dt_dL * dt_dL +
           (d_SagA_rs / (2.0f * r * r * f)) * dr * dr +
           r * (dtheta * dtheta + sinf(theta) * sinf(theta) * dphi * dphi);
    d2.y = -2.0f * dr * dtheta / r + sinf(theta) * cosf(theta) * dphi * dphi;
    d2.z = -2.0f * dr * dphi / r - 2.0f * cosf(theta) / sinf(theta) * dtheta * dphi;
}

// Runge-Kutta 4th order integration step
__device__ void rk4Step(CudaRay& ray, float dL) {
    float3 k1a, k1b, k2a, k2b, k3a, k3b, k4a, k4b;
    
    // Store original values
    float r0 = ray.r, theta0 = ray.theta, phi0 = ray.phi;
    float dr0 = ray.dr, dtheta0 = ray.dtheta, dphi0 = ray.dphi;
    
    // k1
    geodesicRHS(ray, k1a, k1b);
    
    // k2
    ray.r = r0 + 0.5f * dL * k1a.x;
    ray.theta = theta0 + 0.5f * dL * k1a.y;
    ray.phi = phi0 + 0.5f * dL * k1a.z;
    ray.dr = dr0 + 0.5f * dL * k1b.x;
    ray.dtheta = dtheta0 + 0.5f * dL * k1b.y;
    ray.dphi = dphi0 + 0.5f * dL * k1b.z;
    geodesicRHS(ray, k2a, k2b);
    
    // k3
    ray.r = r0 + 0.5f * dL * k2a.x;
    ray.theta = theta0 + 0.5f * dL * k2a.y;
    ray.phi = phi0 + 0.5f * dL * k2a.z;
    ray.dr = dr0 + 0.5f * dL * k2b.x;
    ray.dtheta = dtheta0 + 0.5f * dL * k2b.y;
    ray.dphi = dphi0 + 0.5f * dL * k2b.z;
    geodesicRHS(ray, k3a, k3b);
    
    // k4
    ray.r = r0 + dL * k3a.x;
    ray.theta = theta0 + dL * k3a.y;
    ray.phi = phi0 + dL * k3a.z;
    ray.dr = dr0 + dL * k3b.x;
    ray.dtheta = dtheta0 + dL * k3b.y;
    ray.dphi = dphi0 + dL * k3b.z;
    geodesicRHS(ray, k4a, k4b);
    
    // Final update
    ray.r = r0 + (dL / 6.0f) * (k1a.x + 2.0f * k2a.x + 2.0f * k3a.x + k4a.x);
    ray.theta = theta0 + (dL / 6.0f) * (k1a.y + 2.0f * k2a.y + 2.0f * k3a.y + k4a.y);
    ray.phi = phi0 + (dL / 6.0f) * (k1a.z + 2.0f * k2a.z + 2.0f * k3a.z + k4a.z);
    ray.dr = dr0 + (dL / 6.0f) * (k1b.x + 2.0f * k2b.x + 2.0f * k3b.x + k4b.x);
    ray.dtheta = dtheta0 + (dL / 6.0f) * (k1b.y + 2.0f * k2b.y + 2.0f * k3b.y + k4b.y);
    ray.dphi = dphi0 + (dL / 6.0f) * (k1b.z + 2.0f * k2b.z + 2.0f * k3b.z + k4b.z);
    
    // Update Cartesian coordinates
    ray.x = ray.r * sinf(ray.theta) * cosf(ray.phi);
    ray.y = ray.r * sinf(ray.theta) * sinf(ray.phi);
    ray.z = ray.r * cosf(ray.theta);
}

// Check if ray hits black hole event horizon
__device__ bool interceptBlackHole(const CudaRay& ray) {
    return ray.r <= d_SagA_rs;
}

// Check if ray crosses accretion disk
__device__ bool crossesAccretionDisk(const float3& oldPos, const float3& newPos, 
                                   const AccretionDisk& disk) {
    // Check if ray crosses the equatorial plane (y=0)
    bool crossed = (oldPos.y * newPos.y < 0.0f);
    if (!crossed) return false;
    
    // Check if intersection is within disk bounds
    float r = sqrtf(newPos.x * newPos.x + newPos.z * newPos.z);
    return (r >= disk.innerRadius && r <= disk.outerRadius);
}

// Calculate enhanced accretion disk color with realistic physics
__device__ float4 calculateDiskColor(const float3& position, const AccretionDisk& disk, float time) {
    float r = length_f3(position);
    
    // Temperature decreases with distance (T ∝ r^-0.75 for thin disk)
    float temperature = 50000.0f * powf(d_SagA_rs / (r * 1e10f), 0.75f);
    temperature = fmaxf(2000.0f, fminf(100000.0f, temperature));
    
    // Get blackbody spectrum
    float3 diskColor = blackbodySpectrum(temperature);
    
    // Apply gravitational redshift 
    float redshift = gravitationalRedshift(r);
    diskColor.x *= redshift * 1.5f;
    diskColor.y *= redshift * 1.5f;
    diskColor.z *= redshift * 1.5f;
    
    // Add turbulence and detail using noise
    float noise = sinf(r * 0.0001f + time) * cosf(position.x * 0.0001f) * sinf(position.z * 0.0001f);
    float turbulence = 1.0f + 0.3f * noise;
    diskColor.x *= turbulence;
    diskColor.y *= turbulence;
    diskColor.z *= turbulence;
    
    // Radial brightness falloff
    float radialFalloff = 1.0f - smoothstep(disk.innerRadius, disk.outerRadius, r);
    diskColor.x *= radialFalloff;
    diskColor.y *= radialFalloff;
    diskColor.z *= radialFalloff;
    
    // Add Doppler shifting for rotating disk
    float velocity = sqrtf(d_SagA_rs / r) * 0.3f; // Keplerian velocity approximation
    diskColor = dopplerShift(diskColor, velocity);
    
    return make_float4(diskColor.x, diskColor.y, diskColor.z, 1.0f);
}

// Generate visible light beams
__device__ float3 generateLightBeam(const float3& position, float intensity, float time) {
    float distance = length_f3(position);
    float attenuation = 1.0f / (1.0f + 0.1f * distance * distance / (d_SagA_rs * d_SagA_rs));
    
    // Create animated beam pattern
    float beamIntensity = intensity * attenuation;
    float3 beamColor = make_float3(0.8f, 0.9f, 1.0f);
    beamColor.x *= beamIntensity;
    beamColor.y *= beamIntensity;
    beamColor.z *= beamIntensity;
    
    // Add gravitational lensing effect
    float lensing = 1.0f + d_SagA_rs / distance;
    beamColor.x *= lensing;
    beamColor.y *= lensing;
    beamColor.z *= lensing;
    
    // Add time-based animation
    float pulse = 1.0f + 0.2f * sinf(time * 2.0f);
    beamColor.x *= pulse;
    beamColor.y *= pulse;
    beamColor.z *= pulse;
    
    return beamColor;
}

// Main photorealistic ray tracing kernel
__global__ void photorealisticRaytraceKernel(
    float4* output,
    CudaCamera camera,
    AccretionDisk disk,
    BlackHole blackHole,
    int width, int height,
    float time
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    
    // Calculate ray direction from screen coordinates
    float u = (2.0f * (x + 0.5f) / width - 1.0f) * camera.aspect * camera.tanHalfFov;
    float v = (1.0f - 2.0f * (y + 0.5f) / height) * camera.tanHalfFov;
    
    float3 camPos = make_float3_from_glm(camera.position);
    float3 camRight = make_float3_from_glm(camera.right);
    float3 camUp = make_float3_from_glm(camera.up);
    float3 camForward = make_float3_from_glm(camera.forward);
    
    float3 dir = normalize_f3(make_float3(
        u * camRight.x - v * camUp.x + camForward.x,
        u * camRight.y - v * camUp.y + camForward.y,
        u * camRight.z - v * camUp.z + camForward.z
    ));
    
    // Initialize ray
    CudaRay ray = initRay(camPos, dir);
    
    float4 color = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    float3 prevPos = make_float3(ray.x, ray.y, ray.z);
    float3 lightBeamAccumulation = make_float3(0.0f, 0.0f, 0.0f);
    float timeTravel = 0.0f;
    
    bool hitBlackHole = false;
    bool hitDisk = false;
    
    int maxSteps = camera.moving ? 40000 : 80000;
    
    // Ray marching loop
    for (int step = 0; step < maxSteps; ++step) {
        // Check black hole collision
        if (interceptBlackHole(ray)) {
            hitBlackHole = true;
            break;
        }
        
        // Accumulate gravitational time dilation
        float gravitationalPotential = -d_SagA_rs / (2.0f * ray.r);
        timeTravel += gravitationalPotential * d_D_LAMBDA;
        
        // Add light beam interactions near black hole
        if (ray.r < 5.0f * d_SagA_rs) {
            float beamStrength = expf(-ray.r / d_SagA_rs) * 0.1f;
            float3 beamContrib = generateLightBeam(make_float3(ray.x, ray.y, ray.z), beamStrength, time);
            lightBeamAccumulation.x += beamContrib.x;
            lightBeamAccumulation.y += beamContrib.y;
            lightBeamAccumulation.z += beamContrib.z;
        }
        
        // Integration step
        rk4Step(ray, d_D_LAMBDA);
        
        // Check disk intersection
        float3 newPos = make_float3(ray.x, ray.y, ray.z);
        if (crossesAccretionDisk(prevPos, newPos, disk)) {
            hitDisk = true;
            break;
        }
        
        prevPos = newPos;
        
        // Check escape condition
        if (ray.r > d_ESCAPE_R) break;
    }
    
    // Color calculation based on what was hit
    if (hitDisk) {
        float4 diskColor = calculateDiskColor(make_float3(ray.x, ray.y, ray.z), disk, time);
        
        // Add gravitational lensing brightness enhancement
        float lensing = 1.0f + 2.0f * d_SagA_rs / ray.r;
        diskColor.x *= lensing;
        diskColor.y *= lensing;
        diskColor.z *= lensing;
        
        // Combine with light beams
        diskColor.x += lightBeamAccumulation.x * 0.5f;
        diskColor.y += lightBeamAccumulation.y * 0.5f;
        diskColor.z += lightBeamAccumulation.z * 0.5f;
        
        color = diskColor;
        
    } else if (hitBlackHole) {
        // Enhanced event horizon with Hawking radiation glow
        float lambda = length_f3(make_float3(ray.x - camPos.x, ray.y - camPos.y, ray.z - camPos.z));
        float hawkingGlow = expf(-lambda / (d_SagA_rs * 1000.0f)) * 0.05f;
        
        color = make_float4(
            hawkingGlow * 0.1f + lightBeamAccumulation.x * 0.1f,
            hawkingGlow * 0.05f + lightBeamAccumulation.y * 0.1f,
            hawkingGlow * 0.2f + lightBeamAccumulation.z * 0.1f,
            1.0f
        );
        
    } else {
        // Enhanced background with visible light beams and cosmic background
        float3 background = make_float3(0.01f, 0.01f, 0.03f);
        
        // Add visible light beams in empty space
        background.x += lightBeamAccumulation.x;
        background.y += lightBeamAccumulation.y;
        background.z += lightBeamAccumulation.z;
        
        // Add stars/cosmic background
        float starField = sinf(u * 1000.0f) * cosf(v * 1000.0f);
        if (starField > 0.999f) {
            background.x += 0.3f;
            background.y += 0.27f;
            background.z += 0.24f;
        }
        
        color = make_float4(background.x, background.y, background.z, 1.0f);
    }
    
    // Apply time dilation color effects
    float timeDilationFactor = 1.0f + timeTravel * 0.00001f;
    color.x *= timeDilationFactor;
    color.y *= timeDilationFactor;
    color.z *= timeDilationFactor;
    
    output[idx] = color;
}

// C interface functions
extern "C" {
    void launchPhotorealisticKernel(
        float4* output,
        const CudaCamera& camera,
        const AccretionDisk& disk,
        const BlackHole& blackHole,
        int width, int height,
        float time,
        hipStream_t stream
    ) {
        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                      (height + blockSize.y - 1) / blockSize.y);
        
        photorealisticRaytraceKernel<<<gridSize, blockSize, 0, stream>>>(
            output, camera, disk, blackHole, width, height, time
        );
    }
}